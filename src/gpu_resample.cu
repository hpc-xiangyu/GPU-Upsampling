#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

#include "stdefs.h"
#include "resample.h"

#define CUDA_CHECK_RETURN(value) {                                          \
    hipError_t _m_cudaStat = value;                                        \
    if (_m_cudaStat != hipSuccess) {                                       \
        fprintf(stderr, "Error %s at line %d in file %s\n",                 \
                hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);       \
        exit(1);                                                            \
    } }

WORD 	*d_Vs;
HWORD 	*d_Imp;
HWORD	*d_ImpD;
HWORD	*d_X;
HWORD	*d_TandP, *d_TxorP;
size_t	*d_indices;

HWORD	*TandP, *TxorP;

WORD	*Vs;
size_t	*indices;

const int BLOCKS = 256, THREADS = 256;

__global__ void kernel_FilterUp(HWORD d_Imp[], HWORD d_ImpD[], UHWORD Nwing,  
	HWORD *d_X, HWORD *d_TandP, HWORD *d_TxorP, size_t *d_indices, 
	WORD *d_Vs, int g_count, UHWORD LpScl) {
	/* 1-Dimentional thread blocks */
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	int t_num = blockDim.x * gridDim.x;
	HWORD *Hp, *Hdp = NULL, *End;
	HWORD a = 0;
	WORD t, v = 0;
	HWORD Ph;
	HWORD Inc;

	for (int i = idx; i < g_count; i += t_num) {
		HWORD const *Xp = &d_X[d_indices[i]];
		Inc = 1;
		Ph = d_TandP[i];
		Hp = &d_Imp[Ph>>Na];
		End = &d_Imp[Nwing];
		// Branch-condition: Interp
		Hdp = &d_ImpD[Ph>>Na];
		a = Ph & Amask;
		// Branch-condition: Inc(Right-wing)
		// End--;
		while (Hp < End) {
			t = *Hp;
			t += (((WORD)*Hdp)*a)>>Na;
			Hdp += Npc;
			t *= *Xp;
			if (t & (1<<(Nhxn-1)))
				t += (1<<(Nhxn-1));
			t >>= Nhxn;
			v += t;
			Hp += Npc;
			Xp += Inc;
		}
		d_Vs[i] = v;

		v = 0;
		Ph = d_TxorP[i];
		Hp = &d_Imp[Ph>>Na];
		End = &d_Imp[Nwing];
		// Branch-cond: Interp
		Hdp = &d_Imp[Ph>>Na];
		a = Ph & Amask;
		Xp = &d_X[d_indices[i]] + 1;
		// Branch-condition: Inc(Right-wing)
		End--;
		if (Ph == 0) {
			Hp += Npc;
			Hdp += Npc;
		}
		while (Hp < End) {
			t = *Hp;
			t += (((WORD)*Hdp)*a)>>Na;
			Hdp += Npc;
			t *= *Xp;
			if (t & (1<<(Nhxn-1)))
				t += (1<<(Nhxn-1));
			t >>= Nhxn;
			v += t;
			Hp += Npc;
			Xp += Inc;
		}
		d_Vs[i] += v;
		d_Vs[i] >>= Nhg;
		d_Vs[i] *= LpScl;
	}
}

/* Inlined function defined in resampleisubs.c */
extern "C" HWORD WordToHword(WORD v, int scl);

extern "C" {

void GPU_Init(HWORD Imp[], HWORD ImpD[], UHWORD Nwing, int IBUFFSIZE, int OBUFFSIZE, double factor) {
	/* Imp[] and ImpD[] */
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_Imp, 	sizeof(HWORD)*Nwing));
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_ImpD, 	sizeof(HWORD)*Nwing));
	CUDA_CHECK_RETURN(hipMemcpy(d_Imp, 	Imp, 	sizeof(HWORD)*Nwing, hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(d_ImpD, 	ImpD, 	sizeof(HWORD)*Nwing, hipMemcpyHostToDevice));

	/* X, Y */
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_X,		sizeof(HWORD)*IBUFFSIZE));

	int u_fac = (int)(factor + 1);	
	/* v */
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_Vs,		sizeof(WORD)*IBUFFSIZE*u_fac));
	Vs = (WORD*)malloc(sizeof(WORD)*IBUFFSIZE*u_fac);

	/* TandP TxorP */
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_TandP,	sizeof(HWORD)*IBUFFSIZE*u_fac));
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_TxorP,	sizeof(HWORD)*IBUFFSIZE*u_fac));
	TandP = (HWORD*)malloc(sizeof(HWORD)*IBUFFSIZE*u_fac);
	TxorP = (HWORD*)malloc(sizeof(HWORD)*IBUFFSIZE*u_fac);

	/* *Time>>Np */
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_indices, sizeof(size_t)*IBUFFSIZE*u_fac));
	indices = (size_t*)malloc(sizeof(size_t)*IBUFFSIZE*u_fac);
}

void GPU_Destruct() {
	/* Imp[] and ImpD[] */
	CUDA_CHECK_RETURN(hipFree(d_Imp));	
	CUDA_CHECK_RETURN(hipFree(d_ImpD));

	/* X, Y */
	CUDA_CHECK_RETURN(hipFree(d_X));

	/* v */
	CUDA_CHECK_RETURN(hipFree(d_Vs));

	/* TandP TxorP */
	CUDA_CHECK_RETURN(hipFree(d_TandP));
	CUDA_CHECK_RETURN(hipFree(d_TxorP));
	free(TandP);
	free(TxorP);

	/* *Time>>Np */
	free(indices);
}

int GPU_SrcUP(HWORD X[], HWORD Y[], double factor, UWORD *Time,
			  UHWORD Nx, UHWORD Nwing, UHWORD LpScl,
			  HWORD Imp[], HWORD ImpD[], BOOL Interp, 
			  int IBUFFSIZE, int OBUFFSIZE) {
	WORD v;
	double dt;
	UWORD dtb;

	dt = 1.0/factor;
	dtb = dt*(1<<Np) + 0.5;

	FILE*	fp = fopen("./verify/gpu_TandP", "w");
	
	/* GPU need */
	int g_count = ((1<<Np)*(WORD)Nx + dtb - 1) / dtb;
	int realTime;
	for (int i = 0; i < g_count; ++i) {
		realTime = *Time + i * dtb;
		indices[i] = realTime>>Np;
		TandP[i] = (HWORD)(realTime&Pmask);
		TxorP[i] = (HWORD)(((realTime^Pmask)+1)&Pmask);
		fprintf(fp, "%u\n", TandP[i]);
	}
	*Time += g_count * dtb;

	fclose(fp);

	CUDA_CHECK_RETURN(hipMemcpy(d_indices, indices, sizeof(size_t)*g_count, hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(d_X, X, sizeof(HWORD)*IBUFFSIZE, hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(d_TandP, TandP, sizeof(HWORD)*g_count, hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(d_TxorP, TxorP, sizeof(HWORD)*g_count, hipMemcpyHostToDevice));
	kernel_FilterUp<<<BLOCKS, THREADS>>>(d_Imp, d_ImpD, Nwing, d_X, d_TandP, d_TxorP, d_indices, d_Vs, g_count, LpScl);
	CUDA_CHECK_RETURN(hipMemcpy(Vs, d_Vs, sizeof(WORD)*g_count, hipMemcpyDeviceToDevice));
	for (int i = 0; i < g_count; ++i) 
		Y[i] = WordToHword(v, NLpScl);

	return g_count;
}

}
